#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "mytime.h"

//function to add two elements
//Every function meant for cuda is written by global
__global__
void add(int n, float *x, float *y)
{
    //index =  blockidx.x * blockDim.x + threadIdx.x
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+= stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1<<3; //1M elements
    struct timespec start, end;

    float *x, *y;
    
    //allocate data in unified memory 
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));



    for(int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    //Get device configuration
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // Replace 0 with your device ID if needed

    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max threads per dimension (x, y, z): %d, %d, %d\n", 
            prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max grid size (x, y, z): %d, %d, %d\n", 
            prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);



    //It is very volatile, make sure numBlocks is not in decimals/float. It should be fully divisible, to prevent miscalculation.
    int blockSize = 1024;       //1024 because thats's maximum thread per block
    int numBlocks = (N + blockSize - 1)/blockSize;

    std::cout << N << std::endl << numBlocks << std::endl;

    start = get_timespec();
    //Run on GPU; Launch the kernel to execute the function on GPU
    //add<<<1,256>>>(N, x, y);
    add<<<numBlocks,blockSize>>>(N, x, y);

    //Is the lauched kernel is async, we are waiting here for kerenl to finish it's job.
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();

    if (err != hipSuccess) 
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    end = get_timespec();

    std::cout << "Time Taken(ms): " << diff_timespec_us(start, end)/1000 << std::endl;

    //Check errors, all values are 3.0f
    float max_error = 0.0f;
    for(int i = 0; i < N; i++)
    {
        max_error = fmax(max_error, fabs(y[i]-3.0f));
        std::cout << y[i] << ", ";
    }
    std::cout << std::endl;
    std::cout << "Max Error:" << max_error << std::endl;

    //Free Memory
    hipFree(x);
    hipFree(y);

    return 0;
}