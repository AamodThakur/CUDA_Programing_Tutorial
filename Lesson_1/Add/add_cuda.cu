#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "mytime.h"

//function to add two elements
//Every function meant for cuda is written by global
__global__
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
    {
        y[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1<<25; //1M elements
    struct timespec start, end;

    float *x, *y;
    
    //allocate data in unified memory 
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));



    for(int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    start = get_timespec();
    //Run on GPU; Launch the kernel to execute the function on GPU
    add<<<1,1>>>(N, x, y);

    //Is the lauched kernel is async, we are waiting here for kerenl to finish it's job.
    hipDeviceSynchronize();

    end = get_timespec();

    std::cout << "Time Taken(ms): " << diff_timespec_us(start, end)/1000 << std::endl;

    //Check errors, all values are 3.0f
    float max_error = 0.0f;
    for(int i = 0; i < N; i++)
    {
        max_error = fmax(max_error, fabs(y[i]-3.0f));
    }
    std::cout << "Max Error:" << max_error << std::endl;

    //Free Memory
    hipFree(x);
    hipFree(y);

    return 0;
}